#include "CUDA/CUDAMemoryManager.h"
#include "CUDA/CUDAUtils.h"
// TODO：delelte future
#include <iostream>

CUDAMemoryManager::CUDAMemoryManager(){ }

void* CUDAMemoryManager::allocate(size_t bytes) {
    if (bytes == 0) {
        std::cerr << "Invalid allocation size: 0 bytes." << std::endl;
        exit(-1);
    }
    CUDA_CHECK(hipGetLastError());
    void* devPtr;
    CUDA_CHECK(hipMalloc((void **)&devPtr, bytes));
    return devPtr;
}

void CUDAMemoryManager::deallocate(void* ptr) {
    CUDA_CHECK(hipFree(ptr));
    ptr = nullptr;
}