#include "CUDA/CUDAWorker.h"
#include "CUDA/CUDAUtils.h"
// 删掉
#include "Batch.h"

hipblasHandle_t handle;

CUDAWorker::CUDAWorker(int id, Engine *engine) : Worker(id, engine)
{
    std::cout << "[CUDAWorker " << get_id() << "] Created." << std::endl;
    LinearFuncPtr linear_func = OpRegistry::Instance().Linear().Get(CUDA, FLOAT32);
    model_ = std::make_unique<Linear>(linear_func);
}

CUDAWorker::~CUDAWorker()
{
    stop();
    std::cout << "[CUDAWorker " << get_id() << "] Destroyed." << std::endl;
}

void CUDAWorker::handle_init(Command cmd)
{
    // 1. 获取设备上下文
    // std::cout << "[Worker " << get_id() << " TID: " << get_thread_id_str() << "] Handling INIT command..." << std::endl;
    CUDA_CHECK(hipSetDevice(get_id()));
    is_initialized_ = true;
    // void *d_array = memory_manager_.allocate(1000000 * sizeof(float));
    
    // 2. 装载参数
    // std::this_thread::sleep_for(std::chrono::seconds(500));

    // 3. 传回信号
    try
    {
        Result res;
        res.success = true;
        res.output_data = "Worker " + std::to_string(get_id()) + " initialized successfully.";
        cmd.individual_promise.set_value(res);
    }
    catch (const std::future_error &e)
    {
        std::cerr << "[Worker " << get_id() << " TID: " << get_thread_id_str()
                  << "] Future error setting value for INIT promise: " << e.what() << std::endl;
    }
}

void CUDAWorker::handle_infer(Command cmd)
{
    // 这里应该重写写一下未初始化的逻辑
    // if (!is_initialized_)
    // {
    //     Result res;
    //     res.request_id = cmd.request_id;
    //     res.success = false;
    //     res.error_message = "Worker " + std::to_string(get_id()) + " not initialized for INFER.";
    //     // This worker failed its part. If it's the last one, it sets the master promise.
    //     if (cmd.remaining_workers->fetch_sub(1, std::memory_order_acq_rel) == 1)
    //     {
    //         std::cout << "[Worker " << get_id() << " TID: " << get_thread_id_str() << "] (ReqID: " << cmd.request_id << ") Last worker, setting master promise (due to its own init error)." << std::endl;
    //         cmd.master_promise->set_value(res);
    //     }
    //     else
    //     {
    //         std::cout << "[Worker " << get_id() << " TID: " << get_thread_id_str() << "] (ReqID: " << cmd.request_id << ") Not last worker, error reported for its part." << std::endl;
    //     }
    //     return;
    // }

    // 获取需要处理的输入数据
    std::string partial_output = cmd.input_data;
    Batch bt;
    model_->forward(bt);
    // TODO：进行计算

    // 最后一个设备处理完了 通知 Engine
    if (cmd.remaining_workers->fetch_sub(1, std::memory_order_acq_rel) == 1)
    {
        std::cout << "[Worker " << get_id() << " TID: " << get_thread_id_str() << "] (ReqID: " << cmd.request_id << ") This is the LAST worker for this INFER task. Setting master promise." << std::endl;
        Result final_res;
        final_res.request_id = cmd.request_id;
        final_res.success = true;
        // TODO：这里之后稍微改改
        std::reverse(partial_output.begin(), partial_output.end());
        partial_output = "[W0_reversed] " + partial_output;
        final_res.output_data = "Aggregated (simulated by last worker " + std::to_string(get_id()) + "): " + partial_output;
        cmd.master_promise->set_value(final_res);
    }
    else
    {
        std::cout << "[Worker " << get_id() << " TID: " << get_thread_id_str() << "] (ReqID: " << cmd.request_id << ") Not the last worker for this INFER task. (" << cmd.remaining_workers->load() << " remaining)" << std::endl;
    }
}
