#include "CUDAMemoryManager.h"
#include <hip/hip_runtime.h>
#include <iostream>

CUDAMemoryManager::CUDAMemoryManager(){ }

void* CUDAMemoryManager::allocate(size_t bytes) {
    if (bytes == 0) {
        std::cerr << "Invalid allocation size: 0 bytes." << std::endl;
        exit(-1);
    }

    hipError_t lastErr = hipGetLastError();
    if (lastErr != hipSuccess) {
        std::cerr << "Previous CUDA error: " << hipGetErrorString(lastErr) << std::endl;
        hipDeviceReset();  // 重置设备状态
        exit(-1);
    }

    void* devPtr;
    hipError_t err = hipMalloc((void **)&devPtr, bytes);
    if (err != hipSuccess) {
        std::cerr << "hipMalloc failed! Error: " << hipGetErrorString(err) << std::endl;
        // Handle the error (e.g., return, exit, or clean up resources)
        exit(-1);
    }
    return devPtr;
}

void CUDAMemoryManager::deallocate(void* ptr) {
    hipError_t err = hipFree(ptr);
    if (err != hipSuccess) {
        std::cout << " free ptr: "<< ptr << std::endl;
        std::cerr << "hipFree failed! Error: " << hipGetErrorString(err) << std::endl;
        std::cerr << "CUDA Error Code: " << err << std::endl;
        exit(EXIT_FAILURE);
    }
    ptr = nullptr;
}