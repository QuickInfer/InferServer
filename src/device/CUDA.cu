#include <hip/hip_runtime.h>
#include "allocator/cuda/CUDAAllocator.h"
#include "function/cuda/CUDAFunction.h"

CUDA::CUDA() {
    deviceName = "CUDA";
    allocator = new CUDAAllocator();
    F = new CUDAFunction();
}

// 从 CPU 内存中取数据并传输到设备
void CUDA::move_in(float* ptr_dev, float* ptr_cpu, size_t bytes) {
    hipError_t err = hipMemcpy(ptr_dev, ptr_cpu, bytes, hipMemcpyHostToDevice);
}

// 从设备内存中取数据并传输到 CPU
void CUDA::move_out(float* ptr_dev, float* ptr_cpu, size_t bytes) {
    hipError_t err = hipMemcpy(ptr_cpu, ptr_dev, bytes, hipMemcpyDeviceToHost);
}

// 分配设备内存
void CUDA::allocate(float* ptr, size_t size) {
    ptr = allocator->allocate(size);
}

// 回收设备内存
void CUDA::deallocate(float* ptr) {
    allocator->deallocate(ptr);
}