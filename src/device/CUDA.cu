#include <hip/hip_runtime.h>
#include "CUDA.h"
#include "allocator/cuda/CUDAAllocator.h"
#include "function/cuda/CUDAFunction.h"


CUDA::CUDA() {
    device = "CUDA";
    allocator = new CUDAAllocator();
    F = new CUDAFunction();
}

// 从 CPU 内存中取数据并传输到设备
void CUDA::move_in(float* ptr_dev, float* ptr_cpu, size_t size) {
    // std::cout << "move data into cuda" << std::endl;
    hipError_t err = hipMemcpy(ptr_dev, ptr_cpu, sizeof(float)*size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}

// 从设备内存中取数据并传输到 CPU
void CUDA::move_out(float* ptr_dev, float* ptr_cpu, size_t size) {
    hipError_t err = hipMemcpy(ptr_cpu, ptr_dev, sizeof(float)*size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}

// 分配设备内存
float* CUDA::allocate(size_t size) {
    float* ptr = (float*)allocator->allocate(size*sizeof(float));
    return ptr;
}

// 回收设备内存
void CUDA::deallocate(float* ptr) {
    allocator->deallocate((void*)ptr);
}

void CUDA::copy(float* from, float* to, size_t size) {
    hipError_t err = hipMemcpy(to, from, size * sizeof(float), hipMemcpyDeviceToDevice);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy failed! Error: " << hipGetErrorString(err) << std::endl;
        // 处理错误，例如退出、返回 nullptr 或记录日志
        exit(EXIT_FAILURE);
    }
}