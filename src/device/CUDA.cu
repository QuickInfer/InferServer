#include <hip/hip_runtime.h>
#include "CUDA.h"
#include "allocator/cuda/CUDAAllocator.h"
#include "function/cuda/CUDAFunction.h"


CUDA::CUDA() {
    device = "CUDA";
    allocator = new CUDAAllocator();
    F = new CUDAFunction();
}

CUDA::~CUDA() {
    delete allocator;
    delete F;
}

// 从 CPU 内存中取数据并传输到设备
void CUDA::move_in(void* ptr_dev, void* ptr_cpu, size_t bytes) {
    // std::cout << "move data into cuda" << std::endl;
    hipError_t err = hipMemcpy(ptr_dev, ptr_cpu, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}

// 从设备内存中取数据并传输到 CPU
void CUDA::move_out(void* ptr_dev, void* ptr_cpu, size_t bytes) {
    // 检查指针有效性
    if (ptr_dev == nullptr || ptr_cpu == nullptr) {
        std::cerr << "Invalid pointer!" << std::endl;
        exit(-1);
    }

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "hipDeviceSynchronize failed: " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }

    // 调用 hipMemcpy
    err = hipMemcpy(ptr_cpu, ptr_dev, bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }

    // 检查最后的 CUDA 错误
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Last CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}

// 分配设备内存
void* CUDA::allocate(size_t bytes) {
    return allocator->allocate(bytes);
}

// 回收设备内存
void CUDA::deallocate(void* ptr) {
    allocator->deallocate((void*)ptr);
}

void CUDA::copy(void* dst, void* src, size_t bytes) {
    hipError_t err = hipMemcpy(dst, src, bytes, hipMemcpyDeviceToDevice);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy failed! Error: " << hipGetErrorString(err) << std::endl;
        // 处理错误，例如退出、返回 nullptr 或记录日志
        exit(EXIT_FAILURE);
    }
}