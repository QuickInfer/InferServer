#include "CUDAAllocator.h"
#include <hip/hip_runtime.h>
#include <iostream>

CUDAAllocator::CUDAAllocator(){ }

void* CUDAAllocator::allocate(size_t size) {
    if (size == 0) {
        std::cerr << "Invalid allocation size: 0 bytes." << std::endl;
        exit(-1);
    }

    hipError_t lastErr = hipGetLastError();
    if (lastErr != hipSuccess) {
        std::cerr << "Previous CUDA error: " << hipGetErrorString(lastErr) << std::endl;
        hipDeviceReset();  // 重置设备状态
        exit(-1);
    }

    void* devPtr;
    hipError_t err = hipMalloc((void **)&devPtr, size);
    if (err != hipSuccess) {
        std::cerr << "hipMalloc failed! Error: " << hipGetErrorString(err) << std::endl;
        // Handle the error (e.g., return, exit, or clean up resources)
        exit(-1);
    }
    return devPtr;
}

void CUDAAllocator::deallocate(void* ptr) {
    hipError_t err = hipFree(ptr);
    if (err != hipSuccess) {
        std::cerr << "hipFree failed! Error: " << hipGetErrorString(err) << std::endl;
        // 根据需求处理错误，例如退出程序或记录日志
        exit(EXIT_FAILURE);
    }
    ptr = nullptr;
}