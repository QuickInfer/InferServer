#include "CUDAAllocator.h"
#include <hip/hip_runtime.h>
#include <iostream>

CUDAAllocator::CUDAAllocator(){ }

void* CUDAAllocator::allocate(size_t bytes) {
    if (bytes == 0) {
        std::cerr << "Invalid allocation size: 0 bytes." << std::endl;
        exit(-1);
    }

    hipError_t lastErr = hipGetLastError();
    if (lastErr != hipSuccess) {
        std::cerr << "Previous CUDA error: " << hipGetErrorString(lastErr) << std::endl;
        hipDeviceReset();  // 重置设备状态
        exit(-1);
    }

    void* devPtr;
    hipError_t err = hipMalloc((void **)&devPtr, bytes);
    if (err != hipSuccess) {
        std::cerr << "hipMalloc failed! Error: " << hipGetErrorString(err) << std::endl;
        // Handle the error (e.g., return, exit, or clean up resources)
        exit(-1);
    }
    return devPtr;
}

void CUDAAllocator::deallocate(void* ptr) {
    hipError_t err = hipFree(ptr);
    if (err != hipSuccess) {
        std::cout << " free ptr: "<< ptr << std::endl;
        std::cerr << "hipFree failed! Error: " << hipGetErrorString(err) << std::endl;
        std::cerr << "CUDA Error Code: " << err << std::endl;
        exit(EXIT_FAILURE);
    }
    ptr = nullptr;
}