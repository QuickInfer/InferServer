#include "CUDAAllocator.h"
#include <hip/hip_runtime.h>

CUDAAllocator::CUDAAllocator(){ }

float* CUDAAllocator::allocate(std::size_t size) {
    float* devPtr;
    hipMalloc((void **)&devPtr, size);
    return devPtr;
}

void CUDAAllocator::deallocate(void* ptr) {
    hipFree(ptr);
}