#include "CUDAAllocator.h"
#include <hip/hip_runtime.h>

CUDAAllocator::CUDAAllocator(){ }

void* CUDAAllocator::allocate(size_t size) {
    void* devPtr;
    hipMalloc((void **)&devPtr, size);
    return devPtr;
}

void CUDAAllocator::deallocate(void* ptr) {
    hipFree(ptr);
    ptr = nullptr;
}