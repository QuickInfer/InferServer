#include "CUDAAllocator.h"
#include <hip/hip_runtime.h>

CUDAAllocator::CUDAAllocator(){ }

void CUDAAllocator::allocate(void* devPtr, std::size_t size) {
    hipMalloc((void **)&devPtr, size);
}

void CUDAAllocator::deallocate(void* ptr) {
    hipFree(ptr);
}