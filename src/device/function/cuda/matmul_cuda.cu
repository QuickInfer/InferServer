#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "matmul_cuda.h"
#include "common.h"

// TODO:日后要是有需要的话，给他变成 矩阵乘矩阵吧
//      初始化和销毁的工作应该交给外边运行
#ifdef USE_CUBLAS
#include <hipblas.h>

// 全局的 cuBLAS 句柄，可以在初始化时创建和销毁
static hipblasHandle_t g_cublas_handle = nullptr;

// 初始化 cuBLAS 句柄
void init_cublas() {
    if (g_cublas_handle == nullptr) {
        hipblasCreate(&g_cublas_handle);
    }
}

// 销毁 cuBLAS 句柄
void destroy_cublas() {
    if (g_cublas_handle != nullptr) {
        hipblasDestroy(g_cublas_handle);
        g_cublas_handle = nullptr;
    }
}

void matmul_cuda(float *xout, const float *x, const float *w, int n, int d) {
    // 初始化 cuBLAS 句柄，给外边运行
    // init_cublas();

    // 使用 cuBLAS 进行矩阵向量乘法
    // w (D x N) 矩阵，x (N x 1) 向量，xout (D x 1) 向量
    // cuBLAS 默认使用列主序存储，需要转置矩阵 w
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasStatus_t status = hipblasSgemv(
        g_cublas_handle,
        HIPBLAS_OP_T,   // 对矩阵 w 进行转置
        n,             // 转置前的列数
        d,             // 转置前的行数
        &alpha,
        w,             // 输入矩阵 w
        n,             // 转置后的主维度（即 leading dimension）
        x,             // 输入向量 x
        1,
        &beta,
        xout,          // 输出向量 xout
        1
    );

    if (status != HIPBLAS_STATUS_SUCCESS) {
        // 错误处理
        printf("CUBLAS matmul failed\n");
    }

    // 如果需要，可以在程序结束时销毁 cuBLAS 句柄
    // destroy_cublas();
}
#else
// CUDA 内核实现矩阵乘法
__global__ void matmul_kernel(float *xout,const float *x,const float *w, int n, int d) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= d)
        return;

    float sum = 0.0f;
    for (int j = 0; j < n; j++) {
        sum += w[i * n + j] * x[j];
    }
    xout[i] = sum;
}

void matmul_cuda(float *xout, const float *x, const float *w, int n, int d) {

    // 计算线程块和网格大小
    int blockSize = num_threads_small;
    int gridSize = (d + blockSize - 1) / blockSize;

    // 限制 gridSize，避免过多的线程块
    gridSize = min(gridSize, 1024); // 根据 GPU 的规格调整

    // 调用 CUDA 内核
    matmul_kernel<<<gridSize, blockSize>>>(xout, x, w, n, d);

}
#endif
