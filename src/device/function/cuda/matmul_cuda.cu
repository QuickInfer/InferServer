#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "matmul_cuda.h"
#include "common.h"

// TODO:日后要是有需要的话，给他变成 矩阵乘矩阵吧
//      初始化和销毁的工作应该交给外边运行
#ifdef USE_CUBLAS
#include <hipblas.h>

// 全局的 cuBLAS 句柄，可以在初始化时创建和销毁
static hipblasHandle_t g_cublas_handle = nullptr;

// 初始化 cuBLAS 句柄
void init_cublas() {
    if (g_cublas_handle == nullptr) {
        hipblasCreate(&g_cublas_handle);
    }
}

// 销毁 cuBLAS 句柄
void destroy_cublas() {
    if (g_cublas_handle != nullptr) {
        hipblasDestroy(g_cublas_handle);
        g_cublas_handle = nullptr;
    }
}

void matmul_cuda(float *xout, const float *x, const float *w, int n, int d) {
    // 初始化 cuBLAS 句柄，给外边运行
    // init_cublas();

    // 使用 cuBLAS 进行矩阵向量乘法
    // w (D x N) 矩阵，x (N x 1) 向量，xout (D x 1) 向量
    // cuBLAS 默认使用列主序存储，需要转置矩阵 w
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasStatus_t status = hipblasSgemv(
        g_cublas_handle,
        HIPBLAS_OP_T,   // 对矩阵 w 进行转置
        n,             // 转置前的列数
        d,             // 转置前的行数
        &alpha,
        w,             // 输入矩阵 w
        n,             // 转置后的主维度（即 leading dimension）
        x,             // 输入向量 x
        1,
        &beta,
        xout,          // 输出向量 xout
        1
    );

    if (status != HIPBLAS_STATUS_SUCCESS) {
        // 错误处理
        printf("CUBLAS matmul failed\n");
    }

    // 如果需要，可以在程序结束时销毁 cuBLAS 句柄
    // destroy_cublas();
}
#else
// CUDA 内核实现矩阵乘法
__global__ void matmul_kernel(float *xout, const float *x, const float *w, int n, int d, int batch_size) {
    int batch_idx = blockIdx.y;  // 批处理索引
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // 输出向量索引

    if (i >= d || batch_idx >= batch_size)
        return;

    float sum = 0.0f;
    for (int j = 0; j < n; j++) {
        sum += w[i * n + j] * x[batch_idx * n + j];
    }
    xout[batch_idx * d + i] = sum;
}

void matmul_cuda(float *y, const float *x, const float *w, int n, int d, int batch_size) {

    // 计算线程块和网格大小
    int blockSize = num_threads_small;
    int gridSizeX = (d + blockSize - 1) / blockSize;
    int gridSizeY = batch_size;
    dim3 gridSize(gridSizeX, gridSizeY);

    // 调用 CUDA 内核
    matmul_kernel<<<gridSize, blockSize>>>(y, x, w, n, d, batch_size);

}
#endif
