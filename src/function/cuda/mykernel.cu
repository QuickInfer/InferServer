#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA 核函数
__global__ void addVectors(const float* a, const float* b, float* c, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n)
        c[idx] = a[idx] + b[idx];
}

// 供 C++ 调用的包装函数
extern "C" void addVectorsWrapper(const float* a, const float* b, float* c, int n) {
    // 分配设备内存
    float *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, n * sizeof(float));
    hipMalloc((void**)&d_b, n * sizeof(float));
    hipMalloc((void**)&d_c, n * sizeof(float));

    // 将数据复制到设备
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    // 启动核函数
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    addVectors<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    // 将结果复制回主机
    hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

