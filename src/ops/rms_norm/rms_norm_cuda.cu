#include "hip/hip_runtime.h"
#include "registry.h"
#include "CUDA/CUDAUtils.h"


__global__ void rmsnorm_kernel(float *x, float *w, int n, int batch_size, float epsilon, int elementsPerThread) 
{
    int batch_idx = blockIdx.y;  // 批次索引
    // 计算输入和输出的偏移量
    float *x_batch = x + batch_idx * n;

    float ss = 0.0f;
    for (int i = 0; i < elementsPerThread; i++) {
        int j = threadIdx.x + i * num_threads_large;
        if (j < n)
            ss += x_batch[j] * x_batch[j];
    }

    using BlockReduce = hipcub::BlockReduce<float, num_threads_large>;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    ss = BlockReduce(temp_storage).Sum(ss);

    // 计算归一化因子
    __shared__ float shared_ss;
    if (threadIdx.x == 0) {
        ss /= n;
        ss += epsilon;
        ss = 1.0f / sqrtf(ss);
        shared_ss = ss;
    }
    __syncthreads();
    
    float ss_normalized = shared_ss;

    // 归一化并缩放
    for (int i = 0; i < elementsPerThread; i++) {
        int j = threadIdx.x + i * num_threads_large;
        if (j < n) {
            x_batch[j] = w[j] * (ss_normalized * x_batch[j]);
        }
    }
}

void cuda_fp32_rmsnorm_exec(void *x, void *w, int n, int num, float e)
{
    int elementsPerThread = divUp(n, num_threads_large);
    dim3 blockSize(num_threads_large);
    dim3 gridSize(1, num);

    rmsnorm_kernel<<<gridSize, blockSize>>>((float*)x, (float*)w, n, num, e, elementsPerThread);
}

REGISTER_OP_FUNCTION(RMSNorm, CUDA, FLOAT32, cuda_fp32_rmsnorm_exec);
